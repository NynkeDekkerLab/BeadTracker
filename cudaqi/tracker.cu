
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "tracker.h"

template<typename T> void safeCudaFree(T*& ptr) {
	if (ptr) {
		hipFree(ptr);
		ptr = 0;
	}
}

Tracker::Tracker(uint w, uint h) {
	magic = TRACKER_MAGIC;
	d_buf = 0;
	d_original = 0;
}

Tracker::~Tracker() {
	safeCudaFree(d_buf);
	safeCudaFree(d_original);
}

void Tracker::setImage(uint8_t* data) {
	hipMallocPitch(
}

