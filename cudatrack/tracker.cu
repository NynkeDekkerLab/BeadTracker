#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "tracker.h"
#include "Array2D.h"


template<typename T> void safeCudaFree(T*& ptr) {
	if (ptr) {
		hipFree(ptr);
		ptr = 0;
	}
}

class TrackerBuffer
{
public:
	Array2D<pixel_t, float>* image;
	reducer_buffer<float> reduceBuffer;

	TrackerBuffer(uint w,uint h) : reduceBuffer(w,h) {
		image = new Array2D<pixel_t,float>(w, h);
	}
	~TrackerBuffer()
	{
		if (image) delete image;
	}
};

Tracker::Tracker(uint w, uint h) {
	magic = TRACKER_MAGIC;

	width = w;
	height = h;
	buffer = new TrackerBuffer(w,h);
}

Tracker::~Tracker() {
	delete buffer;
}

void Tracker::setImage(pixel_t* data, uint pitchInBytes) {
	buffer->image->set(data, pitchInBytes);
}


struct TestImgComputePixel {
	float xpos, ypos, S;
	float compute(uint x, uint y) {
/*		if (x==0&&y==0)
			printf("value: %f", value);
*/
		float X = x + 0.5f - xpos;
		float Y = y + 0.5f - ypos;
		float r = sqrtf(X*X+Y*Y)+1;
		float v = sinf( (r-10)*2*3.141593f*S);
		return v*v / (r * r * S);
	}
};


void Tracker::loadTestImage(float xpos, float ypos, float S)
{
	TestImgComputePixel pixel_op = { xpos, ypos, 1.0f/S };

	// generate
	float* buf = new float[width*height];
	for (uint y=0;y<height;y++)
		for(uint x=0;x<width;x++)
			buf[y*width+x] = pixel_op.compute(x,y);

	// normalize
	float minv, maxv;
	minv=maxv=buf[0];
	for (int k=0;k<width*height;k++) {
		minv=std::min(minv, buf[k]);
		maxv=std::max(maxv, buf[k]);
	}
	// convert to uchar
	uchar *ibuf = new uchar[width*height];
	for (int k=0;k<width*height;k++)
		ibuf[k]= 255.0f * (buf[k]-minv)/(maxv-minv);
	delete[] buf;

	buffer->image->set(ibuf, sizeof(pixel_t)*width);
	delete[] ibuf;
}

vector2f Tracker::ComputeCOM()
{
	if (!buffer->image)
		return vector2f();

	vector2f com;
	com.x = buffer->image->momentX(buffer->reduceBuffer);
	com.y = buffer->image->momentY(buffer->reduceBuffer);
	float sum = buffer->image->sum(buffer->reduceBuffer);
	com.x /= sum;
	com.y /= sum;
	return com;
}

vector2f Tracker::XCorLocalize(vector2f initial)
{
	vector2f estimate;

	return initial;
}

void Tracker::copyToHost(pixel_t* data, uint pitchInBytes)
{
	if (buffer->image)
		buffer->image->copyToHost(data, pitchInBytes);
}

void* Tracker::getCurrentBufferImage() {
	return buffer->image;
}
