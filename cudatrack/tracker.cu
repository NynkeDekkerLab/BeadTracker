#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "tracker.h"
#include "Array2D.h"


template<typename T> void safeCudaFree(T*& ptr) {
	if (ptr) {
		hipFree(ptr);
		ptr = 0;
	}
}

class TrackerBuffer
{
public:
	Array2D<pixel_t, float>* image;
	reducer_buffer<float> reduceBuffer;

	TrackerBuffer(uint w,uint h) : reduceBuffer(w,h) {
		image = 0;
	}
	~TrackerBuffer()
	{
		if (image) delete image;
	}
};

template<typename T, typename TC>
static vector2f ComputeCOM(Array2D<T, TC>* image, reducer_buffer<TC>& reduceBuffer)
{
	vector2f com;
	com.x = image->momentX(reduceBuffer);
	com.y = image->momentY(reduceBuffer);
	float sum = image->sum(reduceBuffer);
	com.x /= sum;
	com.y /= sum;
	return com;
}

Tracker::Tracker(uint w, uint h) {
	magic = TRACKER_MAGIC;

	width = w;
	height = h;
	buffer = new TrackerBuffer(w,h);
}

Tracker::~Tracker() {
}

void Tracker::setImage(pixel_t* data, uint pitchInBytes) {
	
}


struct TestImgComputePixel {
	float xpos, ypos, S;
	__device__ __host__ float operator()(float value, uint x, uint y) {
/*		if (x==0&&y==0)
			printf("value: %f", value);
*/
		float X = x + 0.5f - xpos;
		float Y = y + 0.5f - ypos;
		float r = sqrtf(X*X+Y*Y)+1;
		float v = sinf( (r-10)*2*3.141593f*S);
		return v*v / (r * r * S);
	}
};


void Tracker::loadTestImage(float xpos, float ypos, float S)
{
	if (!buffer->image) {
		buffer->image = new Array2D<pixel_t,float>(width, height);
	}
	TestImgComputePixel pixel_op = { xpos, ypos, 1.0f/S };
	buffer->image->applyPerPixel(pixel_op);
	/*float maxValue = buffer->image->maximum(buffer->reduceBuffer);
	float minValue = buffer->image->maximum(buffer->reduceBuffer);
	buffer->image->multiplyAdd(1.0f / (maxValue-minValue), -minValue / (maxValue-minValue ));*/
}

vector2f Tracker::ComputeCOM()
{
	if (!buffer->image)
		return vector2f();

	return ::ComputeCOM(buffer->image, buffer->reduceBuffer);
}

vector2f Tracker::XCorLocalize(vector2f initial)
{
	vector2f estimate;

	return initial;
}

void Tracker::copyToHost(pixel_t* data, uint pitchInBytes)
{
	if (buffer->image)
		buffer->image->copyToHost(data, pitchInBytes);
}

void* Tracker::getCurrentBufferImage() {
	return buffer->image;
}
