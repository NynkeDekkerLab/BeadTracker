
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "tracker.h"

template<typename T> void safeCudaFree(T*& ptr) {
	if (ptr) {
		hipFree(ptr);
		ptr = 0;
	}
}

Tracker::Tracker(uint w, uint h) {
	magic = TRACKER_MAGIC;
}

Tracker::~Tracker() {
}

void Tracker::setImage(uchar* data) {

}

